#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

int main()
{
    const int N = 1024;              // Array size
    std::vector<float> h_A(N, 1.0f); // Host array A
    std::vector<float> h_B(N, 2.0f); // Host array B
    std::vector<float> h_C(N, 0.0f); // Host array C

    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_A, h_A.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), N * sizeof(float), hipMemcpyHostToDevice);

    // Load PTX module
    hipModule_t module;
    
    hipError_t err = hipInit(0);
    if (err != hipSuccess)
    {
        const char *errMsg;
        hipDrvGetErrorString(err, &errMsg);
        std::cerr << "CUDA Initialization Error: " << errMsg << std::endl;
        return EXIT_FAILURE;
    }

    if (hipModuleLoad(&module, "/home/huanqi/projects/cuda_study/ptx/add.ptx") != hipSuccess)
    {
        std::cerr << "Failed to load PTX module" << std::endl;
        return EXIT_FAILURE;
    }

    // Get kernel function
    hipFunction_t vector_add;
    hipModuleGetFunction(&vector_add, module, "vector_add");

    // Set up kernel arguments
    void *args[] = {&d_A, &d_B, &d_C, (void*)&N};

    // Launch kernel
    int threads_per_block = 256;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    hipModuleLaunchKernel(vector_add,
                   blocks_per_grid, 1, 1,
                   threads_per_block, 1, 1,
                   0, 0, args, 0);

    // Copy result back to host
    hipMemcpy(h_C.data(), d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    hipModuleUnload(module);
    for (int i = 0; i < N; ++i)
    {
        std::cout << h_C[i] << " ";
    }
    // Verify result
    for (int i = 0; i < N; ++i)
    {
        if (h_C[i] != h_A[i] + h_B[i])
        {
            std::cerr << "Verification failed at index " << i << "\n";
            return EXIT_FAILURE;
        }
    }

    std::cout << "Verification passed!\n";
    return EXIT_SUCCESS;
}